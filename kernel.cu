#include "hip/hip_runtime.h"
#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) CUT_BANK_CHECKER(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) CUT_BANK_CHECKER(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif


__global__ void executeFirstLayer(float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU)
{
        int blockID=blockIdx.x;
        int pixelX=threadIdx.x;
        int pixelY=threadIdx.y;

    int kernelTemplate[25] = {
        0,  1,  2,  3,  4,
        29, 30, 31, 32, 33,
        58, 59, 60, 61, 62,
        87, 88, 89, 90, 91,
        116,117,118,119,120 };


        int weightBegin=blockID*26;
        int windowX=pixelX*2;
        int windowY=pixelY*2;

        float result=0;

        result+=Layer1_Weights_GPU[weightBegin];

        ++weightBegin;

        for(int i=0;i<25;++i)
        {
                result+=Layer1_Neurons_GPU[windowY*29+windowX+kernelTemplate[i]]*Layer1_Weights_GPU[weightBegin+i];
        }

        result=(1.7159*tanhf(0.66666667*result));

        Layer2_Neurons_GPU[13*13*blockID+pixelY*13+pixelX]=result;

}

layer1CPU {

for (int j = 0; j < 6; j++) {
    for (int i =0; i < 26; i++) {
        int result = 0;
        
        if (i == 0) {
            result += Layer1_Weights_CPU[j*26];
        } else {
            for (int k = 0; k < 13; k++) {
                for (int l = 0; l < 13; l++) {
                    result += Layer1_Neurons_CPU[k * 2 * 29 + l * 2 + kernelTemplate[i - 1] * Layer1_Weights_CPU[j * 26 + i - 1];
                }
            }
        }
        
        result=(1.7159*tanhf(0.66666667*result));
        Layer2_Neurons_CPU[13 * 13 * j + k * 2 * 13 + l * 2] = result;
    }
}

}



__global__ void executeSecondLayer(float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU)
{
        int blockID=blockIdx.x;
        int pixelX=threadIdx.x;
        int pixelY=threadIdx.y;

        int kernelTemplate2[25] = {
        0,  1,  2,  3,  4,
        13, 14, 15, 16, 17,
        26, 27, 28, 29, 30,
        39, 40, 41, 42, 43,
        52, 53, 54, 55, 56   };

        int weightBegin=blockID*26*6;
        int windowX=pixelX*2;
        int windowY=pixelY*2;

        float result=0;


        result+=Layer2_Weights_GPU[weightBegin];

        if(blockID==1 && pixelX==0 && pixelY==0)
        {
                result+=0;
        }

        ++weightBegin;

        for (int i=0; i<25; ++i )
    {
        result+=Layer2_Neurons_GPU[windowX + 13*windowY +kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6];
        result+=Layer2_Neurons_GPU[169 + windowX + 13*windowY +kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+1];
                result+=Layer2_Neurons_GPU[338 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+2];
        result+=Layer2_Neurons_GPU[507 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+3];
        result+=Layer2_Neurons_GPU[676 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+4];
        result+=Layer2_Neurons_GPU[845 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+5];
        }

        result=(1.7159*tanhf(0.66666667*result));

        Layer3_Neurons_GPU[5*5*blockID+pixelY*5+pixelX]=result;
}

__global__ void executeThirdLayer(float *Layer3_Neurons_GPU, float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU)
{
        int blockID=blockIdx.x;
        int pixelY=threadIdx.y;


        int weightBegin=blockID*1251;

        float result=0;

        result+=Layer3_Weights_GPU[weightBegin];

        ++weightBegin;

    for (int i=0; i<1250; ++i )
    {
                result+=Layer3_Neurons_GPU[i]*Layer3_Weights_GPU[weightBegin+i];
    }

        result=(1.7159*tanhf(0.66666667*result));

        Layer4_Neurons_GPU[blockID]=result;

}

__global__ void executeFourthLayer(float *Layer4_Neurons_GPU,float *Layer4_Weights_GPU,float *Layer5_Neurons_GPU)
{
        int blockID=blockIdx.x;
        int pixelY=threadIdx.y;


        int weightBegin=blockID*101;

        float result=0;

        result+=Layer4_Weights_GPU[weightBegin];

        ++weightBegin;

    for (int i=0; i<100; ++i )
    {
                result+=Layer4_Neurons_GPU[i]*Layer4_Weights_GPU[weightBegin+i];
    }

        result=(1.7159*tanhf(0.66666667*result));
        //printf("layer5%f\n", result);
        Layer5_Neurons_GPU[blockID]=result;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
