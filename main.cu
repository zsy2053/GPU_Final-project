#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

// includes, project
//#include <cutil.h>
#include <hip/hip_vector_types.h>

// includes, kernels
#include </nfs/ug/homes-1/h/huangq38/NVIDIA_CUDA-7.0_Samples/00_mine/project/NN_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);
void NeuralNetwork();

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
        //printf("Start!");
        NeuralNetwork();
    //CUT_EXIT(argc, argv);
}

void InitGPUMem(float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU,float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU,float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU,float *Layer4_Weights_GPU,float *Layer5_Neurons_GPU)
{
    hipError_t err = hipSuccess;

        err = hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(float)*29*29);
        err = hipMalloc((void**) &Layer1_Weights_GPU, sizeof(float)*156);

        err = hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(float)*13*13*6);
        err = hipMalloc((void**) &Layer2_Weights_GPU, sizeof(float)*7800);

        err = hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(float)*1250);
        err = hipMalloc((void**) &Layer3_Weights_GPU, sizeof(float)*125100);

        err = hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(float)*100);
        err = hipMalloc((void**) &Layer4_Weights_GPU, sizeof(float)*1010);

        err = hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(float)*10);
}
void InitHostMem(float *Layer1_Weights_CPU,float *Layer2_Weights_CPU,float *Layer3_Weights_CPU,float *Layer4_Weights_CPU)
{
        // initial layer 1 weight
        FILE * pFile1 = fopen ("lw1.wei","rb");
        if (pFile1 != NULL)
        {
        for(int i=0;i<156;++i)
                fread(&(Layer1_Weights_CPU[i]),sizeof(float),1,pFile1);
                fclose (pFile1);
        }

        // initial layer 2 weight
        FILE * pFile2 = fopen ("lw2.wei","rb");
        if (pFile2 != NULL)
        {
                fread(Layer2_Weights_CPU,sizeof(float),7800,pFile2);
                fclose (pFile2);
        }
        // initial layer 3 weight
        FILE * pFile3 = fopen ("lw3.wei","rb");
        if (pFile3 != NULL)
        {
                fread(Layer3_Weights_CPU,sizeof(float),125100,pFile3);
                fclose (pFile3);
        }
        // initial layer 4 weight
        FILE * pFile4 = fopen ("lw4.wei","rb");
        if (pFile4 != NULL)
        {
                fread(Layer4_Weights_CPU,sizeof(float),1010,pFile4);
                fclose (pFile4);
        }
}

void readIn(float *layer1)
{
        FILE *fp;
        fp=fopen("in.neu","rb");
        if(fp)
        {
                fread(layer1,sizeof(float),29*29,fp);
                fclose(fp);
        }
}

void output(double *final)
{
        FILE *fp=0;
        fp=fopen("out.res","wb");
        if(fp)
        {
                fwrite(final,sizeof(double),10,fp);
                fclose(fp);
        }
}

void NeuralNetwork()
{
        //CUT_DEVICE_INIT();

float Layer1_Neurons_CPU[29*29]=
{1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,0,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,0,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,0,0,1,0,1,0,0,1,0,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,0,0,1,0,1,0,0,0,0,0,0,0,0,0,0,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};
//readIn(Layer1_Neurons_CPU);

        float *Layer1_Neurons_GPU;
        float Layer1_Weights_CPU[156];
        float *Layer1_Weights_GPU;

        float Layer2_Weights_CPU[7800];
        float *Layer2_Weights_GPU;
        float *Layer2_Neurons_GPU;

        float Layer3_Weights_CPU[125100];
        float *Layer3_Weights_GPU;
        float *Layer3_Neurons_GPU;

        float Layer4_Weights_CPU[1010];
        float *Layer4_Weights_GPU;
        float *Layer4_Neurons_GPU;

        float Layer5_Neurons_CPU[10]={0,0,0,0,0,0,0,0,0,0};
        float *Layer5_Neurons_GPU;

        double *outputLayer;
        unsigned int timer = 0;
        float totaltime = 0.0f;
        //init input here
        InitHostMem(Layer1_Weights_CPU,Layer2_Weights_CPU,Layer3_Weights_CPU,Layer4_Weights_CPU);


        //allocate momory on Device
        InitGPUMem(Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer2_Neurons_GPU,Layer2_Weights_GPU,Layer3_Neurons_GPU,Layer3_Weights_GPU,Layer4_Neurons_GPU,Layer4_Weights_GPU,Layer5_Neurons_GPU);
        hipError_t err = hipSuccess;

        err = hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(float)*29*29);
        err = hipMalloc((void**) &Layer1_Weights_GPU, sizeof(float)*156);

        err = hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(float)*13*13*6);
        err = hipMalloc((void**) &Layer2_Weights_GPU, sizeof(float)*7800);

        err = hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(float)*1250);
        err = hipMalloc((void**) &Layer3_Weights_GPU, sizeof(float)*125100);

        err = hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(float)*100);
        err = hipMalloc((void**) &Layer4_Weights_GPU, sizeof(float)*1010);

        err = hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(float)*10);

        outputLayer = (double*)malloc(sizeof(double)*29*29);
        //init 29x29 handwritting array
        // already done in "initial"

        //copy from CPU to GPU
        err = hipMemcpy(Layer1_Neurons_GPU,Layer1_Neurons_CPU, sizeof(float)*29*29, hipMemcpyHostToDevice);
        err = hipMemcpy(Layer1_Weights_GPU,Layer1_Weights_CPU, sizeof(float)*156, hipMemcpyHostToDevice);
        err = hipMemcpy(Layer2_Weights_GPU,Layer2_Weights_CPU, sizeof(float)*7800, hipMemcpyHostToDevice);
        err = hipMemcpy(Layer3_Weights_GPU,Layer3_Weights_CPU, sizeof(float)*125100, hipMemcpyHostToDevice);
        err = hipMemcpy(Layer4_Weights_GPU,Layer4_Weights_CPU, sizeof(float)*1010, hipMemcpyHostToDevice);
    err = hipMemcpy(Layer5_Neurons_GPU,Layer5_Neurons_CPU, sizeof(float)*10, hipMemcpyHostToDevice);

        // CUT_SAFE_CALL(cutCreateTimer(&timer));
        // CUT_SAFE_CALL(cutStartTimer(timer));

        dim3 Layer1_Block(6,1);
        dim3 Layer1_Thread(13,13);
        executeFirstLayer<<<Layer1_Block,Layer1_Thread>>>(Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer2_Neurons_GPU);

        dim3 Layer2_Block(50,1);
        dim3 Layer2_Thread(5,5);
        executeSecondLayer<<<Layer2_Block,Layer2_Thread>>>(Layer2_Neurons_GPU, Layer2_Weights_GPU,Layer3_Neurons_GPU);

        dim3 Layer3_Block(100,1);
        dim3 Layer3_Thread(1,1);
        executeThirdLayer<<<Layer3_Block,Layer3_Thread>>>(Layer3_Neurons_GPU, Layer3_Weights_GPU,Layer4_Neurons_GPU);

        dim3 Layer4_Block(10,1);
        dim3 Layer4_Thread(1,1);
        executeFourthLayer<<<Layer4_Block,Layer4_Thread>>>(Layer4_Neurons_GPU,Layer4_Weights_GPU,Layer5_Neurons_GPU);

        //*CUT_CHECK_ERROR("Kernel execution failed");

  //  CUT_SAFE_CALL(cutStopTimer(timer));

//      totaltime = cutGetTimerValue(timer);

        //copy from GPU to CPU
    err = hipMemcpy(Layer5_Neurons_CPU,Layer5_Neurons_GPU, sizeof(float)*10, hipMemcpyDeviceToHost);

    // stop and destroy timer

    //printf("Processing time: %f (ms) \n", totaltime);
        //  CUT_SAFE_CALL(cutDeleteTimer(timer));

        for(int a=0;a<10;a++)
        {
                outputLayer[a] = (double)Layer5_Neurons_CPU[a];
                printf("%f\n", outputLayer[a]);
                //if ((a + 1) % 29 == 0){
                //printf("%.0f, \n", abs(outputLayer[a]));
                //} else {
                //printf("%.0f,",abs(outputLayer[a]));
                //}
        }
        //output(outputLayer);


        float Layer4_Neurons_CPU[100];
        err = hipMemcpy(Layer4_Neurons_CPU,Layer4_Neurons_GPU,sizeof(float)*100,hipMemcpyDeviceToHost);
        FILE *fp=fopen("layer_4.neu","wb");
        fwrite(Layer4_Neurons_CPU,sizeof(float),100,fp);
        fclose(fp);

        float Layer3_Neurons_CPU[50*5*5];
        err = hipMemcpy(Layer3_Neurons_CPU,Layer3_Neurons_GPU,sizeof(float)*50*5*5,hipMemcpyDeviceToHost);
        fp=fopen("layer_3.neu","wb");
        fwrite(Layer3_Neurons_CPU,sizeof(float),50*5*5,fp);
        fclose(fp);

        float Layer2_Neurons_CPU[13*13*6];
        err = hipMemcpy(Layer2_Neurons_CPU,Layer2_Neurons_GPU,sizeof(float)*13*13*6,hipMemcpyDeviceToHost);
        fp=fopen("layer_2.neu","wb");
        fwrite(Layer2_Neurons_CPU,sizeof(float),13*13*6,fp);
        fclose(fp);

        fp=fopen("layer_1.neu","wb");
        fwrite(Layer1_Neurons_CPU,sizeof(float),29*29,fp);
        fclose(fp);

        //exit(0);
}
