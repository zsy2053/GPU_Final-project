#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define AS(i, j) CUT_BANK_CHECKER(((float*)&As[0][0]), (BLOCK_SIZE * i + j))
#define BS(i, j) CUT_BANK_CHECKER(((float*)&Bs[0][0]), (BLOCK_SIZE * i + j))
#else
#define AS(i, j) As[i][j]
#define BS(i, j) Bs[i][j]
#endif


__global__ void executeFirstLayer(float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU)
{
        __shared__ float max[512];
	int blockID=blockIdx.x;
	int pixelX=threadIdx.x;
	int pixelY=threadIdx.y;
	

    int kernelTemplate[25] = {
        0,  1,  2,  3,  4,
        29, 30, 31, 32, 33,
        58, 59, 60, 61, 62,
        87, 88, 89, 90, 91,
        116,117,118,119,120 };


	int weightBegin=blockID*26;
	int windowX=pixelX*2;
	int windowY=pixelY*2;
	max[windowX] = Layer1_Weights_GPU[windowX];
	__syncthreads();

	float result=0;

	result+=Layer1_Weights_GPU[weightBegin];

	++weightBegin;

	for(int i=0;i<25;++i)
	{
		result+=Layer1_Neurons_GPU[windowY*29+windowX+kernelTemplate[i]]*Layer1_Weights_GPU[weightBegin+i];
	}

	result=(1.7159*tanhf(0.66666667*result));

	Layer2_Neurons_GPU[13*13*blockID+pixelY*13+pixelX]=result;

}

__global__ void executeSecondLayer(float *Layer2_Neurons_GPU, float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU)
{
        __shared__ float max[512];
	int blockID=blockIdx.x;
	int pixelX=threadIdx.x;
	int pixelY=threadIdx.y;

	int kernelTemplate2[25] = {
        0,  1,  2,  3,  4,
        13, 14, 15, 16, 17, 
        26, 27, 28, 29, 30,
        39, 40, 41, 42, 43, 
        52, 53, 54, 55, 56   };

	int weightBegin=blockID*26*6;
	int windowX=pixelX*2;
	int windowY=pixelY*2;
	max[windowX] = Layer2_Weights_GPU[windowX];
	__syncthreads();
    
	float result=0;

	
	result+=Layer2_Weights_GPU[weightBegin];
	
	if(blockID==1 && pixelX==0 && pixelY==0)
	{
		result+=0;
	}

	++weightBegin;

	for (int i=0; i<25; ++i )
    {
        result+=Layer2_Neurons_GPU[windowX + 13*windowY +kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6];
        result+=Layer2_Neurons_GPU[169 + windowX + 13*windowY +kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+1];
	result+=Layer2_Neurons_GPU[338 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+2];
        result+=Layer2_Neurons_GPU[507 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+3];
        result+=Layer2_Neurons_GPU[676 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+4];
        result+=Layer2_Neurons_GPU[845 + windowX + 13*windowY + kernelTemplate2[i]]*Layer2_Weights_GPU[weightBegin+i*6+5];
	}

	result=(1.7159*tanhf(0.66666667*result));

	Layer3_Neurons_GPU[5*5*blockID+pixelY*5+pixelX]=result;
}

__global__ void executeThirdLayer(float *Layer3_Neurons_GPU, float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU)
{
        __shared__ float max[512];
	int blockID=blockIdx.x;
	int pixelY=threadIdx.y;


	int weightBegin=blockID*1251;
	max[weightBegin] = Layer3_Weights_GPU[weightBegin];
        __syncthreads();

	float result=0;

	result+=Layer3_Weights_GPU[weightBegin];

	++weightBegin;

    for (int i=0; i<1250; ++i )
    {
		result+=Layer3_Neurons_GPU[i]*Layer3_Weights_GPU[weightBegin+i];
    }

	result=(1.7159*tanhf(0.66666667*result));

	Layer4_Neurons_GPU[blockID]=result;

}

__global__ void executeFourthLayer(float *Layer4_Neurons_GPU,float *Layer4_Weights_GPU,float *Layer5_Neurons_GPU)
{
        __shared__ float max[512];
	int blockID=blockIdx.x;
	int pixelY=threadIdx.y;


	int weightBegin=blockID*101;
	max[weightBegin] = Layer4_Weights_GPU[weightBegin];
        __syncthreads();
 
	float result=0;

	result+=Layer4_Weights_GPU[weightBegin];

	++weightBegin;

    for (int i=0; i<100; ++i )
    {
		result+=Layer4_Neurons_GPU[i]*Layer4_Weights_GPU[weightBegin+i];
    }

	result=(1.7159*tanhf(0.66666667*result));

	Layer5_Neurons_GPU[blockID]=result;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
