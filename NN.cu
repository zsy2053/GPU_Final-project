// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <NN_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);
void NeuralNetwork();

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	NeuralNetwork();
}

void InitGPUMem(float *Layer1_Neurons_GPU,float *Layer1_Weights_GPU,float *Layer2_Neurons_GPU,float *Layer2_Weights_GPU,float *Layer3_Neurons_GPU,float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU,float *Layer4_Weights_GPU,float *Layer5_Neurons_GPU)
{
	hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(float)*29*29);
	hipMalloc((void**) &Layer1_Weights_GPU, sizeof(float)*156);
	
	hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(float)*13*13*6);
	hipMalloc((void**) &Layer2_Weights_GPU, sizeof(float)*7800);

	hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(float)*1250);
	hipMalloc((void**) &Layer3_Weights_GPU, sizeof(float)*125100);

	hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(float)*100);
	hipMalloc((void**) &Layer4_Weights_GPU, sizeof(float)*1010);

	hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(float)*10);
}
void InitHostMem(float *Layer1_Weights_CPU,float *Layer2_Weights_CPU,float *Layer3_Weights_CPU,float *Layer4_Weights_CPU)
{
	// initial layer 1 weight
	FILE * pFile1 = fopen ("lw1.wei","rb");
	if (pFile1 != NULL)
	{
	for(int i=0;i<156;++i)
		fread(&(Layer1_Weights_CPU[i]),sizeof(float),1,pFile1);
		fclose (pFile1);
	}

	// initial layer 2 weight
	FILE * pFile2 = fopen ("lw2.wei","rb");
	if (pFile2 != NULL)
	{
		fread(Layer2_Weights_CPU,sizeof(float),7800,pFile2);
		fclose (pFile2);
	}
	// initial layer 3 weight
	FILE * pFile3 = fopen ("lw3.wei","rb");
	if (pFile3 != NULL)
	{
		fread(Layer3_Weights_CPU,sizeof(float),125100,pFile3);
		fclose (pFile3);
	}
	// initial layer 4 weight
	FILE * pFile4 = fopen ("lw4.wei","rb");
	if (pFile4 != NULL)
	{
		fread(Layer4_Weights_CPU,sizeof(float),1010,pFile4);
		fclose (pFile4);
	}
}

void readIn(float *layer1)
{
	FILE *fp;
	fp=fopen("in.neu","rb");
	if(fp)
	{
		fread(layer1,sizeof(float),29*29,fp);
		fclose(fp);
	}
}

void output(double *final)
{
	FILE *fp=0;
	fp=fopen("out.res","wb");
	if(fp)
	{
		fwrite(final,sizeof(double),10,fp);
		fclose(fp);
	}
}

void NeuralNetwork()
{
	float Layer1_Neurons_CPU[29*29]={1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,0,0,0,1,1,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,0,0,0,0,0,0,0,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,0,0,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,
1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};

//readIn(Layer1_Neurons_CPU);

	float *Layer1_Neurons_GPU;
	float Layer1_Weights_CPU[156];
	float *Layer1_Weights_GPU;

	float Layer2_Weights_CPU[7800];
	float *Layer2_Weights_GPU;
	float *Layer2_Neurons_GPU;

	float Layer3_Weights_CPU[125100];
	float *Layer3_Weights_GPU;
	float *Layer3_Neurons_GPU;

	float Layer4_Weights_CPU[1010];
	float *Layer4_Weights_GPU;
	float *Layer4_Neurons_GPU;

	float Layer5_Neurons_CPU[10]={0,0,0,0,0,0,0,0,0,0};
	float *Layer5_Neurons_GPU;

	double *outputLayer;
	unsigned int timer = 0;
	float totaltime = 0.0f;
	//init input here
	InitHostMem(Layer1_Weights_CPU,Layer2_Weights_CPU,Layer3_Weights_CPU,Layer4_Weights_CPU);


	//allocate momory on Device
	//InitGPUMem(Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer2_Neurons_GPU,Layer2_Weights_GPU,Layer3_Neurons_GPU,Layer3_Weights_GPU,Layer4_Neurons_GPU,Layer4_Weights_GPU,Layer5_Neurons_GPU);
	hipMalloc((void**) &Layer1_Neurons_GPU, sizeof(float)*29*29);
	hipMalloc((void**) &Layer1_Weights_GPU, sizeof(float)*156);
	
	hipMalloc((void**) &Layer2_Neurons_GPU, sizeof(float)*13*13*6);
	hipMalloc((void**) &Layer2_Weights_GPU, sizeof(float)*7800);

	hipMalloc((void**) &Layer3_Neurons_GPU, sizeof(float)*1250);
	hipMalloc((void**) &Layer3_Weights_GPU, sizeof(float)*125100);

	hipMalloc((void**) &Layer4_Neurons_GPU, sizeof(float)*100);
	hipMalloc((void**) &Layer4_Weights_GPU, sizeof(float)*1010);

	hipMalloc((void**) &Layer5_Neurons_GPU, sizeof(float)*10);
	outputLayer = (double*)malloc(sizeof(double)*10);
	//init 29x29 handwritting array
	// already done in "initial"

	//copy from CPU to GPU
	hipMemcpy(Layer1_Neurons_GPU,Layer1_Neurons_CPU, sizeof(float)*29*29, hipMemcpyHostToDevice);
	hipMemcpy(Layer1_Weights_GPU,Layer1_Weights_CPU, sizeof(float)*156, hipMemcpyHostToDevice);
	hipMemcpy(Layer2_Weights_GPU,Layer2_Weights_CPU, sizeof(float)*7800, hipMemcpyHostToDevice);
	hipMemcpy(Layer3_Weights_GPU,Layer3_Weights_CPU, sizeof(float)*125100, hipMemcpyHostToDevice);
	hipMemcpy(Layer4_Weights_GPU,Layer4_Weights_CPU, sizeof(float)*1010, hipMemcpyHostToDevice);
	hipMemcpy(Layer5_Neurons_GPU,Layer5_Neurons_CPU, sizeof(float)*10, hipMemcpyHostToDevice);

	// CUT_SAFE_CALL(cutCreateTimer(&timer));
	// CUT_SAFE_CALL(cutStartTimer(timer));

	dim3 Layer1_Block(6,1);
	dim3 Layer1_Thread(13,13);
	executeFirstLayer<<<Layer1_Block,Layer1_Thread>>>(Layer1_Neurons_GPU,Layer1_Weights_GPU,Layer2_Neurons_GPU);

	dim3 Layer2_Block(50,1);
	dim3 Layer2_Thread(5,5);
	executeSecondLayer<<<Layer2_Block,Layer2_Thread>>>(Layer2_Neurons_GPU, Layer2_Weights_GPU,Layer3_Neurons_GPU);

	dim3 Layer3_Block(100,1);
	dim3 Layer3_Thread(1,1);
	executeThirdLayer<<<Layer3_Block,Layer3_Thread>>>(Layer3_Neurons_GPU, Layer3_Weights_GPU,Layer4_Neurons_GPU);

	dim3 Layer4_Block(10,1);
	dim3 Layer4_Thread(1,1);
	executeFourthLayer<<<Layer4_Block,Layer4_Thread>>>(Layer4_Neurons_GPU,Layer4_Weights_GPU,Layer5_Neurons_GPU);

	


//	totaltime = cutGetTimerValue(timer);

	//copy from GPU to CPU
   hipMemcpy(Layer5_Neurons_CPU,Layer5_Neurons_GPU, sizeof(float)*10, hipMemcpyDeviceToHost);
    
    // stop and destroy timer

    //printf("Processing time: %f (ms) \n", totaltime);
	//  CUT_SAFE_CALL(cutDeleteTimer(timer));

	for(int a=0;a<10;a++)
	{
		outputLayer[a] = (double)Layer5_Neurons_CPU[a];
	}
	output(outputLayer);


	float Layer4_Neurons_CPU[100];
	hipMemcpy(Layer4_Neurons_CPU,Layer4_Neurons_GPU,sizeof(float)*100,hipMemcpyDeviceToHost);
	FILE *fp=fopen("layer_4.neu","wb");
	fwrite(Layer4_Neurons_CPU,sizeof(float),100,fp);
	fclose(fp);

	float Layer3_Neurons_CPU[50*5*5];
	hipMemcpy(Layer3_Neurons_CPU,Layer3_Neurons_GPU,sizeof(float)*50*5*5,hipMemcpyDeviceToHost);
	fp=fopen("layer_3.neu","wb");
	fwrite(Layer3_Neurons_CPU,sizeof(float),50*5*5,fp);
	fclose(fp);

	float Layer2_Neurons_CPU[13*13*6];
	hipMemcpy(Layer2_Neurons_CPU,Layer2_Neurons_GPU,sizeof(float)*13*13*6,hipMemcpyDeviceToHost);
	fp=fopen("layer_2.neu","wb");
	fwrite(Layer2_Neurons_CPU,sizeof(float),13*13*6,fp);
	fclose(fp);

	fp=fopen("layer_1.neu","wb");
	fwrite(Layer1_Neurons_CPU,sizeof(float),29*29,fp);
	fclose(fp);

	exit(0);
}